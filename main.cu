#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>

#define FN_NAME(a) (a)
#define LOAD(a) __ldg(a)

#include "stencil_kernels.h"

#undef FN_NAME
#undef LOAD


#define FN_NAME(name) name_ldg
#define LOAD(a) a

#include "stencil_kernels.h"

#undef FN_NAME
#undef LOAD


template<typename T> 
void launch() {

    const size_t isize=256;
    const size_t jsize=256;
    const size_t ksize=60;
    const size_t halo=2;
    const size_t alignment=32;
    const size_t right_padding=isize%alignment;
    const size_t first_padding=alignment-halo;
    const size_t total_size=first_padding+(isize+right_padding)*jsize*ksize;
    const size_t jstride = (isize+right_padding);
    const size_t kstride = jstride*jsize;

    T* a;
    T* b;
    hipMallocManaged(&a, sizeof(T)*total_size);
    hipMallocManaged(&b, sizeof(T)*total_size);    

    const size_t block_size_x = 32;
    const size_t block_size_y = 8;

    const size_t nbx = isize/block_size_x;
    const size_t nby = jsize/block_size_y;
   
    dim3 gd(nbx, nby,1);
    dim3 bd(block_size_x, block_size_y);

    printf("calling copy");

    std::vector<double> timings(5);

    std::chrono::high_resolution_clock::time_point t1,t2;

    for(size_t t=0; t < 10; t++) {
   
        t1 = std::chrono::high_resolution_clock::now();
        copy<<<bd, gd>>>(a,b, first_padding, jstride, kstride);
        hipDeviceSynchronize();
        t2 = std::chrono::high_resolution_clock::now();
        timings[0] += std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count();

        if(!t) {
            for(size_t i=0; i < isize; ++i) {
                for(size_t j=0; j < jsize; ++j) {
                    for(size_t k=0; k < ksize; ++k) {
                        if( b[i+j*jstride + first_padding] != a[i+j*jstride + first_padding] ) {
                            printf("Error in (%d,%d,%d) : %f %f\n", (int)i,(int)j,(int)k,b[i+j*jstride + first_padding], a[i+j*jstride + first_padding]);
                        }
                    }
                }
            }
        }   

        t1 = std::chrono::high_resolution_clock::now();
        delta<<<bd, gd>>>(a,b, first_padding, jstride, kstride);
        t2 = std::chrono::high_resolution_clock::now();
        timings[1] += std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count();

        for(size_t i=0; i < isize; ++i) {
            for(size_t j=0; j < jsize; ++j) {
                for(size_t k=0; k < ksize; ++k) {
                    if( b[i+j*jstride + first_padding] != a[i+j*jstride + first_padding] ) {
                       printf("Error in (%d,%d,%d) : %f %f\n", (int)i,(int)j,(int)k,b[i+j*jstride + first_padding], a[i+j*jstride + first_padding]);
                    }
                }
            }
        }          
    }

//    t1 = std::chrono::high_resolution_clock::now();
    

}

int main(int argc, char** argv) {

    launch<float>();

}
